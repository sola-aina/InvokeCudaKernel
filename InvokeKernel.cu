#include "InvokeKernel.h"
#include "Unique.hpp"

////////////////////////////////////////////////////////////////////////////////

template<typename ...Args> 
void KernelWrapper<void(Args...)>::Invoke( void kernel(Args...)  , int numBlocks , int numThreads , Args ... args )
{
	kernel<<<numBlocks,numThreads>>>( args ... );
    hipDeviceSynchronize();     
}

////////////////////////////////////////////////////////////////////////////////

// Creates 'list' of kernel types
#undef DEFINE_KERNEL
#define DEFINE_KERNEL( Kernel ) ,decltype(Kernel)
#define CREATE_TYPELIST #define TypeList KERNEL_REFERENCES
CREATE_TYPELIST

// Explicitly instantiate kernel if unique
#undef DEFINE_KERNEL
#define DEFINE_KERNEL( Kernel ) template struct KernelWrapper<UniqueType<__COUNTER__ TypeList>>;
KERNEL_REFERENCES

////////////////////////////////////////////////////////////////////////////////

