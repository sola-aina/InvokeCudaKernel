// Copyright (c) 2018 Sola Aina
// 
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
// 
// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.
// 
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

#include "InvokeKernel.h"
#include "Unique.hpp"

////////////////////////////////////////////////////////////////////////////////

template<typename ...Args> 
void KernelWrapper<void(Args...)>::Invoke( void kernel(Args...)  , int numBlocks , int numThreads , Args ... args )
{
	kernel<<<numBlocks,numThreads>>>( args ... );
    hipDeviceSynchronize();     
}

////////////////////////////////////////////////////////////////////////////////

// Creates 'list' of kernel types
#undef DEFINE_KERNEL
#define DEFINE_KERNEL( Kernel ) ,decltype(Kernel)
#define CREATE_TYPELIST #define TypeList KERNEL_REFERENCES
CREATE_TYPELIST

// Explicitly instantiate kernel if unique
#undef DEFINE_KERNEL
#define DEFINE_KERNEL( Kernel ) template struct KernelWrapper<UniqueType<__COUNTER__ TypeList>>;
KERNEL_REFERENCES

////////////////////////////////////////////////////////////////////////////////

